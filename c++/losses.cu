#include "hip/hip_runtime.h"
#include "matrix.h"
#include "losses.h"

/* Loss::calculateLoss()
* -----
* Inherited function, averages the loss calculated 
* from the loss function. 
*
* @y_true: the true labels
* @y_pred: the predicted labels, i.e. output from the last layer, 
*
* Returns: the calculated loss
*/
double Loss::calculateLoss(matrix::Matrix<double> y_true, 
	    matrix::Matrix<double> y_pred) {
    return matrix::sum(loss(y_true, y_pred), 1, false)[0] / y_true.rows; 
}

/* CategoricalCrossEntropy::loss()
* -----
* Calculates the loss w.r.t each sample. 
*
* @y_true: the true labels, expects one-hot
* @y_pred: the predicted labels, i.e. output from the last layer.
*
* Returns: the calculated loss for each sample
*/
matrix::Matrix<double> CategoricalCrossentropy::loss(matrix::Matrix<double> y_true, 
	    matrix::Matrix<double> y_pred) {
    // Expects y_true to be one hot encoded
    return matrix::mul_const(matrix::log(matrix::sum(matrix::mul(y_true, 
			y_pred))), (double) -1.0);
}

/* SparseCategoricalCrossEntropy::loss()
* -----
* Not currently implemented. Calculates the loss w.r.t each sample. 
*
* @y_true: the true labels, expects 1D array of integers
* @y_pred: the predicted labels, i.e. output from the last layer.
*
* Returns: the calculated loss for each sample
*/
matrix::Matrix<double> SparseCategoricalCrossentropy::loss(matrix::Matrix<double> y_true, 
	    matrix::Matrix<double> y_pred) {
    // NEEDS TO BE -np.log(y_pred[range(len(y_true)), y_true]):wq 
    return matrix::mul_const(matrix::log(y_true), (double) -1.0);
}
