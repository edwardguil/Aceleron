/*
* Author: Edward Guilfoyle
* Note: This file demonstrates PARALLEL computing to perform classification on spiral dataset.
*/
#include <iomanip>
#include <chrono>
#include "matrix.h"
#include "losses.h"
#include "layers.h"
#include "metrics.h"
#include "optimizers.h"
#include "hip/hip_runtime.h"
#include "data/data.h"

using namespace matrix;

void handle_input(Matrix<double, double*>& x_train, Matrix<double, double*>& y_train, 
		Matrix<double, double*>& x_test, Matrix<double, double*>& y_test, int N);


int handle_N(int N);

/* main()
* -----
* The main entry point for this program. Creates and runs 
* an Artificial Neural Network. Architecture constists of: 
* 			- Dense(2, 16)
* 			- ReLU()
* 			- Dense(16, 2)
* 			- SoftMax()
* 
* A single command line argument can be parsed to control the
* size of the input data to the neural network. Please pass
* integers only.
*
* @argc: The length of argv
* @argv: Inputs from the command line
*
* Returns: 1 if the program failed, 0 if successful.
*/
int main(int argc, char *argv[]) {
	// Some code to handle command line input
	int N = argc > 1 ? handle_N(std::stoi(argv[1])) : 1000;
	std::cout << "N: " << N << std::endl;
	int train_size = N * 0.8;

	// Lets define our training and test matrices
    Matrix<double, double*> x_train(train_size, 2);
    Matrix<double, double*> y_train(train_size, 2);
    Matrix<double, double*> x_test(N - train_size, 2);
    Matrix<double, double*> y_test(N - train_size, 2);
	// Load our data in
	handle_input(x_train, y_train, x_test, y_test, N);

	// Define our network
	Dense<double, double*> layer1(2, 16);
	ReLU<double, double*> layer2;
	Dense<double, double*> layer3(16, 2);
	SoftmaxCrossEntropy<double, double*> layer4;
	optimizer::SGD<double, double*> sgd(1.0, 0.001);

	// Main algorithimic loop
    for (int i = 0; i < 2001; i++) {
		// Lets do a forward pass
		Matrix<double, double*> out1 = layer1.forward(x_train);
		Matrix<double, double*> out2 = layer2.forward(out1);
		Matrix<double, double*> out3 = layer3.forward(out2);
		Matrix<double, double*> out4 = layer4.forward(out3, y_train);
		
		// Lets calculate some metrics
		double loss = layer4.get_loss();
		double acc = metric::accuracy(y_train, out4);

		// Lets do a backward pass
		Matrix<double, double*> back4 = layer4.backward(out4, y_train);
		Matrix<double, double*> back3 = layer3.backward(out2, back4);
		Matrix<double, double*> back2 = layer2.backward(out1, back3);
		Matrix<double, double*> back1 = layer1.backward(x_train, back2);
		
		// Lets update the model parameters with SGD
		sgd.pre_update();
		sgd.update(&layer3);
		sgd.update(&layer1);
		sgd.post_update();
		_free();	
		if (i % 100 == 0) {
			// Lets test our network every 100 iterations
			Matrix<double, double*> outtest1 = layer1.forward(x_test);
			Matrix<double, double*> outtest2 = layer2.forward(outtest1);
			Matrix<double, double*> outtest3 = layer3.forward(outtest2);
			Matrix<double, double*> outtest4 = layer4.forward(outtest3, y_test);
			
			double losstest = layer4.get_loss();
			double acctest = metric::accuracy(y_test, outtest4);

			std::cout <<  "CUDA   - ";
			std::cout << "epoch: " << i;
			std::cout << ", acc: " << std::setprecision(3) << acc;
			std::cout << ", loss: " << std::setprecision(3) << loss;
			std::cout << ", acc_test: " << std::setprecision(3) << acctest;
			std::cout << ", loss_test: " << std::setprecision(3) << losstest;
			std::cout << ", lr: " << std::fixed << std::setprecision(3) << sgd.get_lr() << std::endl;
			_free();	
		}

    }
	return 0;
}

/* handle_input()
* -----
* Selects which data should be used in training the neural network.
* Selected conditionally on the size of N.
*
* @x_train: The training data samples
* @y_train: The training data labels
* @x_test: The test data samples
* @y_test: The test data labels
* @N: The amount of data that should be inputted into the Matrix's
*/
void handle_input(Matrix<double, double*>& x_train, Matrix<double, double*>& y_train, 
		Matrix<double, double*>& x_test, Matrix<double, double*>& y_test, int N) {
	if (N == 100) {
		x_train.set_matrix(&(x_train_raw_100[0]));
		y_train.set_matrix(&(y_train_raw_100[0]));
		x_test.set_matrix(&(x_test_raw_100[0]));
		y_test.set_matrix(&(y_test_raw_100[0]));
	} else if (N == 200)   {
		x_train.set_matrix(&(x_train_raw_200[0]));
		y_train.set_matrix(&(y_train_raw_200[0]));
		x_test.set_matrix(&(x_test_raw_200[0]));
		y_test.set_matrix(&(y_test_raw_200[0]));
	} else if (N == 300)   {
		x_train.set_matrix(&(x_train_raw_300[0]));
		y_train.set_matrix(&(y_train_raw_300[0]));
		x_test.set_matrix(&(x_test_raw_300[0]));
		y_test.set_matrix(&(y_test_raw_300[0]));
	} else if (N == 400)  {
		x_train.set_matrix(&(x_train_raw_400[0]));
		y_train.set_matrix(&(y_train_raw_400[0]));
		x_test.set_matrix(&(x_test_raw_400[0]));
		y_test.set_matrix(&(y_test_raw_400[0]));
	} else if (N == 500) {
		x_train.set_matrix(&(x_train_raw_500[0]));
		y_train.set_matrix(&(y_train_raw_500[0]));
		x_test.set_matrix(&(x_test_raw_500[0]));
		y_test.set_matrix(&(y_test_raw_500[0]));
	} else if (N == 600) {
		x_train.set_matrix(&(x_train_raw_600[0]));
		y_train.set_matrix(&(y_train_raw_600[0]));
		x_test.set_matrix(&(x_test_raw_600[0]));
		y_test.set_matrix(&(y_test_raw_600[0]));
	} else if (N == 700) {
		x_train.set_matrix(&(x_train_raw_700[0]));
		y_train.set_matrix(&(y_train_raw_700[0]));
		x_test.set_matrix(&(x_test_raw_700[0]));
		y_test.set_matrix(&(y_test_raw_700[0]));
	} else if (N == 800)  {
		x_train.set_matrix(&(x_train_raw_800[0]));
		y_train.set_matrix(&(y_train_raw_800[0]));
		x_test.set_matrix(&(x_test_raw_800[0]));
		y_test.set_matrix(&(y_test_raw_800[0]));
	} else if (N == 900) {
		x_train.set_matrix(&(x_train_raw_900[0]));
		y_train.set_matrix(&(y_train_raw_900[0]));
		x_test.set_matrix(&(x_test_raw_900[0]));
		y_test.set_matrix(&(y_test_raw_900[0]));
	} else if (N == 1000) {
		x_train.set_matrix(&(x_train_raw_1000[0]));
		y_train.set_matrix(&(y_train_raw_1000[0]));
		x_test.set_matrix(&(x_test_raw_1000[0]));
		y_test.set_matrix(&(y_test_raw_1000[0]));
	} else if (N == 2000) {
		x_train.set_matrix(&(x_train_raw_2000[0]));
		y_train.set_matrix(&(y_train_raw_2000[0]));
		x_test.set_matrix(&(x_test_raw_2000[0]));
		y_test.set_matrix(&(y_test_raw_2000[0]));
	} else if (N == 5000) {
		x_train.set_matrix(&(x_train_raw_5000[0]));
		y_train.set_matrix(&(y_train_raw_5000[0]));
		x_test.set_matrix(&(x_test_raw_5000[0]));
		y_test.set_matrix(&(y_test_raw_5000[0]));
	} else if (N == 10000) {
		x_train.set_matrix(&(x_train_raw_10000[0]));
		y_train.set_matrix(&(y_train_raw_10000[0]));
		x_test.set_matrix(&(x_test_raw_10000[0]));
		y_test.set_matrix(&(y_test_raw_10000[0]));
	} else if (N == 20000) {
		x_train.set_matrix(&(x_train_raw_20000[0]));
		y_train.set_matrix(&(y_train_raw_20000[0]));
		x_test.set_matrix(&(x_test_raw_20000[0]));
		y_test.set_matrix(&(y_test_raw_20000[0]));
	} else if (N == 40000) {
		x_train.set_matrix(&(x_train_raw_40000[0]));
		y_train.set_matrix(&(y_train_raw_40000[0]));
		x_test.set_matrix(&(x_test_raw_40000[0]));
		y_test.set_matrix(&(y_test_raw_40000[0]));
	} else if (N == 60000) {
		x_train.set_matrix(&(x_train_raw_60000[0]));
		y_train.set_matrix(&(y_train_raw_60000[0]));
		x_test.set_matrix(&(x_test_raw_60000[0]));
		y_test.set_matrix(&(y_test_raw_60000[0]));
	} else {
		x_train.set_matrix(&(x_train_raw_80000[0]));
		y_train.set_matrix(&(y_train_raw_80000[0]));
		x_test.set_matrix(&(x_test_raw_80000[0]));
		y_test.set_matrix(&(y_test_raw_80000[0]));
	}
}

/* handle_N()
* -----
* Determines the new size of N conditionally on the size of N.
*
* Returns: the new size of N
*/
int handle_N(int N) {
	if (N <= 100) {
		return 100;
	} else if (N <= 200)   {
		return 200;
	} else if (N <= 300)   {
		return 300;
	} else if (N <= 400)  {
		return 400;
	} else if (N <= 500) {
		return 500;
	} else if (N <= 600) {
		return 600;
	} else if (N <= 700) {
		return 700;
	} else if (N <= 800)  {
		return 800;
	} else if (N <= 900) {
		return 900;
	} else if (N <= 1000) {
		return 1000;
	} else if (N <= 2000) {
		return 2000;
	} else if (N <= 5000) {
		return 5000;
	} else if (N <= 10000) {
		return 10000;
	} else if (N <= 20000) {
		return 20000;
	} else if (N <= 40000) {
		return 40000;
	} else if (N <= 60000) {
		return 60000;
	} else  {
		return 80000;
	}
}
